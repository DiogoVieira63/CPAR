#include "hip/hip_runtime.h"
#include "stencil.h"
#include <printf.h>
#include <time.h>

#define NUM_BLOCKS 512
#define NUM_THREADS_PER_BLOCK 256
#define SIZE NUM_BLOCKS *NUM_THREADS_PER_BLOCK

#define N 10000000

using namespace std;

#define X(i) (i * 2)
#define Y(i) (i * 2 + 1)

#define K 4

float *pointsX, *pointsY, *centroids;
int *cluster;


/*
__global__
void stencilKernel (float *a, float *c) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	// initialise the array with the results
	c[id] = 0;

	// iterate through the neighbours required to calculate
	// the values for the current position of c
	for (int n = id-2; n <= id+2; n++) {
		if (n > 0 && n < 	) c[id]+= a[n];
	}

}
*/

// Function to calculate the euclidean distance
__device__ float distance(float x, float y, float *cl)
{
	return ((cl[0] - x) * (cl[0] - x) + (cl[1] - y) * (cl[1] - y));
}

/*
__global__ void assignClusterKernel(int N, float *a, float *b, int *c)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
	{
		float lowest = distance(vectorX[(i)], vectorY[(i)], &centroid[X(0)]);
		int index_low = 0;
		for (int k = 1; k < K; k++)
		{
			float dist = distance(vectorX[(i)], vectorY[(i)], &centroid[X(k)]);
			if (dist < lowest)
			{
				lowest = dist;
				index_low = k;
			}
		}
		cluster[i] = index_low;
	}
}

__global__ void sumCentroidsKernel(int N, int *size, float *sumX, float *sumY, float *vectorX, float *vectorY)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
	{
		int index_low = cluster[i];
		sumX += vectorX[(i)];
		sumY += vectorY[(i)];
		size[index_low] += 1;
	}
}

__global__ void calculateCentroidsKernel(int N, int *size, float *sumX, float *sumY, float *vectorX, float *vectorY)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
	{
		int index_low = cluster[i];
		sumX += vectorX[(i)];
		sumY += vectorY[(i)];
		size[index_low] += 1;
	}
}

*/

//__global__ void assignCluster(float *pointX, float *pointY, float *centroids, int *size, float* sum )

__global__ void kmeans(float *pointX, float *pointY, float *centroids,unsigned int *size, float *sum)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Print some debugging information
	if (tid >= N)
		return;
	
	//centroids to shared memory
	__shared__ float sharedCentroids[K * 2];
	if (threadIdx.x < K * 2)
	{
		sharedCentroids[threadIdx.x] = centroids[threadIdx.x];
	}
	__syncthreads();


	// Calculate the distance between the point and each centroid
	float min_distance = distance(pointX[tid], pointY[tid], &sharedCentroids[0]);
	int min_index = 0;
	for (int i = 1; i < K; i++)
	{
		float dist = distance(pointX[tid], pointY[tid], &sharedCentroids[2 * i]);
		if (dist < min_distance)
		{
			min_distance = dist;
			min_index = i;
		}
	}

	// Update the sums and sizes of the cluster using shared memory
	atomicAdd(&sum[X(min_index)], pointX[tid]);
	atomicAdd(&sum[Y(min_index)], pointY[tid]);
	atomicAdd(&size[min_index], 1);
}


__global__ void kmeans2(float *pointX, float *pointY, float *centroids,int *cluster)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Print some debugging information
	if (tid >= N)
		return;
	
	//centroids to shared memory
	__shared__ float sharedCentroids[K * 2];
	if (threadIdx.x < K * 2)
	{
		sharedCentroids[threadIdx.x] = centroids[threadIdx.x];
	}
	__syncthreads();


	// Calculate the distance between the point and each centroid
	float min_distance = distance(pointX[tid], pointY[tid], &sharedCentroids[0]);
	int min_index = 0;
	for (int i = 1; i < K; i++)
	{
		float dist = distance(pointX[tid], pointY[tid], &sharedCentroids[2 * i]);
		if (dist < min_distance)
		{
			min_distance = dist;
			min_index = i;
		}
	}
	cluster[tid] = min_index;

}

void launchStencilKernel()
{
	// pointers to the device memory
	float *dX, *dY, *dC;
	//unsigned int *dSize;
	int *dCluster;

	// declare variable with size of the array in bytes
	int bytes = N * sizeof(float);
	//int bytesInt = K * sizeof(unsigned int);
	int bytesCentroids = K * 2 * sizeof(float);
	int bytesCluster = N * sizeof(int);

	// allocate the memory on the device
	printf("Allocating On Device\n");
	hipMalloc((void **)&dX, bytes);
	hipMalloc((void **)&dY, bytes);
	hipMalloc((void **)&dC, bytesCentroids);
	//hipMalloc((void **)&dSize, bytesInt);
	//hipMalloc((void **)&dSum, bytesCentroids);
	hipMalloc((void **)&dCluster, bytesCluster);

	checkCUDAError("mem allocation");

	// copy inputs to the device
	hipMemcpy(dX, pointsX, bytes, hipMemcpyHostToDevice);
	hipMemcpy(dY, pointsY, bytes, hipMemcpyHostToDevice);
	hipMemcpy(dC, centroids, bytesCentroids, hipMemcpyHostToDevice);

	checkCUDAError("memcpy h->d");

	// launch the kernel
	startKernelTime();

	dim3 threads_block(NUM_THREADS_PER_BLOCK);
	int blocks_int = (N + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;
	dim3 blocks(blocks_int);
	printf("Launching with %d blocks of %d threads\n", blocks_int, NUM_THREADS_PER_BLOCK);
	int size[K] = {0};
	for (int i = 0; i <= 20; i++)
	{
		// unsigned int size[K] = {0};
		// float sum[K * 2] = {0};
		// hipMemcpy(dSize, size, bytesInt, hipMemcpyHostToDevice);
		// hipMemcpy(dSum, sum, bytesCentroids, hipMemcpyHostToDevice);
		//kmeans<<<blocks, threads_block>>>(dX, dY, dC, dSize, dSum);
		kmeans2<<<blocks, threads_block>>>(dX, dY, dC, dCluster);

		hipMemcpy(cluster,dCluster, bytesCluster, hipMemcpyDeviceToHost);
		memset(size, 0,K*sizeof(int));
		float sum[K * 2] = {0};

		for (int i = 0; i < N; i++){
			sum[X(cluster[i])] += pointsX[i];
			sum[Y(cluster[i])] += pointsY[i];
			size[cluster[i]]++;
		}
		//hipMemcpy(centroids, dC, bytesCentroids, hipMemcpyDeviceToHost);
		//hipMemcpy(size, dSize, bytesInt, hipMemcpyDeviceToHost);
		//hipMemcpy(sum, dSum, bytesCentroids, hipMemcpyDeviceToHost);

		for (int i = 0; i < K; i++){
			centroids[X(i)] = sum[X(i)] / size[i];
			centroids[Y(i)] = sum[Y(i)] / size[i];
		}
		hipMemcpy(dC, centroids, bytesCentroids, hipMemcpyHostToDevice);
	}

	stopKernelTime();
	checkCUDAError("kernel invocation");
	//int size[K];

	// copy the output to the host
	//hipMemcpy(centroids, dC, bytesCentroids, hipMemcpyDeviceToHost);
	//hipMemcpy(size, dSize, bytesInt, hipMemcpyDeviceToHost);
	checkCUDAError("memcpy d->h");

	// print info about centroids and size
	for (int i = 0; i < K; i++)
	{
		printf("Centroid %d: (%f, %f) with size %d\n", i, centroids[X(i)], centroids[Y(i)], size[i]);
	}

	// free the device memory
	hipFree(dX);
	hipFree(dY);
	hipFree(dC);
	hipFree(dCluster);
	//hipFree(dSize);
	//hipFree(dSum);
	checkCUDAError("mem free");
}


int main(int argc, char **argv)
{
	clock_t start, end;
    start = clock();
    double time_used;
	printf("Starting K-means\n");
	// arrays on the host
	
	pointsX = (float *)malloc(N * sizeof(float));
	pointsY = (float *)malloc(N * sizeof(float));
	centroids = (float *)malloc(K * 2 * sizeof(float));
	cluster = (int *)malloc(N * sizeof(int));

	srand(10);

	printf("Initialising arrays with random values between 0 and 1\n");
	// initialises the array
	for (int i = 0; i < N; ++i)
	{
		pointsX[i] = (float)rand() / RAND_MAX;
		pointsY[i] = (float)rand() / RAND_MAX;
	}

	// initialises the centroids
	printf("Initialising centroids with vectors first K values\n");
	for (int i = 0; i < K; ++i)
	{
		centroids[X(i)] = pointsX[i];
		centroids[Y(i)] = pointsY[i];
	}
	launchStencilKernel();
	end = clock();
    time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Execution time: %lf seconds\n", time_used);
	return 0;
}
